#include "hip/hip_runtime.h"
#include "pycugrape.h"
#include <stdio.h>
#include <stdlib.h>
#include <thrust/complex.h>
#include <math.h>
typedef thrust::complex<R> C;

#define idxctrls(t, nctrl) ((nctrl) + NCTRLS*(t) )
const int ctrls_size = NCTRLS*PLEN;
#define idxstates0(ct, nstate, t, nrow, ri) ((ri) + 2*((nrow) + 46*((t) + (PLEN+1)*((nstate) + NSTATE*(ct)))))
#define psi_out0(t, ri) states[idxstates0(ct, nstate, t, nrow, ri)]
#define psi_out_ct0(t, ri) states[idxstates0(1-(ct), nstate, PLEN-(t), nrow, ri)]
#define idxstates1(ct, nstate, t, nrow, ri) ((ri) + 2*((nrow) + 48*((t) + (PLEN+1)*((nstate) + NSTATE*(ct)))))
#define psi_out1(t, ri) states[idxstates1(ct, nstate, t, nrow, ri)]
#define psi_out_ct1(t, ri) states[idxstates1(1-(ct), nstate, PLEN-(t), nrow, ri)]
#define idxstates2(ct, nstate, t, nrow, ri) ((ri) + 2*((nrow) + 50*((t) + (PLEN+1)*((nstate) + NSTATE*(ct)))))
#define psi_out2(t, ri) states[idxstates2(ct, nstate, t, nrow, ri)]
#define psi_out_ct2(t, ri) states[idxstates2(1-(ct), nstate, PLEN-(t), nrow, ri)]
#define idxstates3(ct, nstate, t, nrow, ri) ((ri) + 2*((nrow) + 52*((t) + (PLEN+1)*((nstate) + NSTATE*(ct)))))
#define psi_out3(t, ri) states[idxstates3(ct, nstate, t, nrow, ri)]
#define psi_out_ct3(t, ri) states[idxstates3(1-(ct), nstate, PLEN-(t), nrow, ri)]

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


__device__ double atomicAddD(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}

R *ctrls_d; // [PLEN][NCTRLS];


R *states_d0;
R *ovlp_r_d0;
R *ovlp_i_d0;
R *d_ovlps_r_d0;
R *d_ovlps_i_d0;

__global__
void prop_state_kernel0_noct(R *ctrls, R *states)
{
    const unsigned int ct = 0;
    const unsigned int nstate = blockIdx.y;
    const unsigned int nrow = threadIdx.x;
    __shared__ C s_psik[2][46];
    __syncthreads();
    const short s = ct ? -1 : 1;

    int nrow_cur = nrow;
    const int i_dst0 = nrow_cur % 23;
    nrow_cur /= 23;
    const int i_dst1 = nrow_cur % 2;
    nrow_cur /= 2;

    /* R psi_out_v = psi_out(0, ri); */
    R psi_out_v_r = psi_out0(0, 0);
    R psi_out_v_i = psi_out0(0, 1);
    for (int t = 1; t <= PLEN; t++) {
        int ctrl_t = ct ? (PLEN - t) : (t-1);
        int idx = 0;
        s_psik[0][nrow] = C::complex(psi_out_v_r, psi_out_v_i);

        for (int k = 1; k <= TAYLOR_ORDER; k++) {
            // psi_k -> (pf*H)psi_k
            // psi_out -> psi_out + (pf*H)psi_k
            __syncthreads();

            // TODO: Parallelize this
            R cpf, pf;
            C ppf, t_ppf, H_psi_k_sum;
            int i_src, src_row, valid;
            H_psi_k_sum = 0;
                    cpf = 1.0 / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -1.73415914478e-05);
                            ppf *= i_dst0 * i_dst0;
                            ppf *= i_dst1 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, -0.774088429845);
                            ppf *= i_dst1 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, 0.774088429845);
                            ppf *= i_dst1 * i_dst1;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, 0.0126591104295);
                            ppf *= i_dst0 ;
                            ppf *= i_dst1 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, -8.16814089933e-06);
                            ppf *= i_dst0 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, 8.16814089933e-06);
                            ppf *= i_dst0 * i_dst0;
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - 0;
                        src_row = src_row * 23 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 0)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - -1;
                        src_row = src_row * 23 + i_src;
                            valid = valid && (i_src < 23);
                                pf *= sqrt((R) (i_src - 0));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - 1;
                        src_row = src_row * 23 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 1)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0125663706144, 0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - -1;
                        src_row = src_row * 23 + i_src;
                            valid = valid && (i_src < 23);
                                pf *= sqrt((R) (i_src - 0));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(-0.0125663706144, -0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - 1;
                        src_row = src_row * 23 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 2)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - -1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src < 2);
                                pf *= sqrt((R) (i_src - 0));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 23 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 23 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 3)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0125663706144, 0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - -1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src < 2);
                                pf *= sqrt((R) (i_src - 0));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 23 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(-0.0125663706144, -0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 23 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
            s_psik[1-idx][nrow] = H_psi_k_sum;
            psi_out_v_r += H_psi_k_sum.real();
            psi_out_v_i += H_psi_k_sum.imag();

            // swap psi_k and psi_k_next
            idx = 1 - idx;
        }

        psi_out0(t, 0) = psi_out_v_r;
        psi_out0(t, 1) = psi_out_v_i;
    }
}
__global__
void prop_state_kernel0_withct(R *ctrls, R *states)
{
    const unsigned int ct = 1;
    const unsigned int nstate = blockIdx.y;
    const unsigned int nrow = threadIdx.x;
    __shared__ C s_psik[2][46];
    __syncthreads();
    const short s = ct ? -1 : 1;

    int nrow_cur = nrow;
    const int i_dst0 = nrow_cur % 23;
    nrow_cur /= 23;
    const int i_dst1 = nrow_cur % 2;
    nrow_cur /= 2;

    /* R psi_out_v = psi_out(0, ri); */
    R psi_out_v_r = psi_out0(0, 0);
    R psi_out_v_i = psi_out0(0, 1);
    for (int t = 1; t <= PLEN; t++) {
        int ctrl_t = ct ? (PLEN - t) : (t-1);
        int idx = 0;
        s_psik[0][nrow] = C::complex(psi_out_v_r, psi_out_v_i);

        for (int k = 1; k <= TAYLOR_ORDER; k++) {
            // psi_k -> (pf*H)psi_k
            // psi_out -> psi_out + (pf*H)psi_k
            __syncthreads();

            // TODO: Parallelize this
            R cpf, pf;
            C ppf, t_ppf, H_psi_k_sum;
            int i_src, src_row, valid;
            H_psi_k_sum = 0;
                    cpf = 1.0 / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -1.73415914478e-05);
                            ppf *= i_dst0 * i_dst0;
                            ppf *= i_dst1 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, -0.774088429845);
                            ppf *= i_dst1 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, 0.774088429845);
                            ppf *= i_dst1 * i_dst1;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, 0.0126591104295);
                            ppf *= i_dst0 ;
                            ppf *= i_dst1 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, -8.16814089933e-06);
                            ppf *= i_dst0 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, 8.16814089933e-06);
                            ppf *= i_dst0 * i_dst0;
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - 0;
                        src_row = src_row * 23 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 0)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - -1;
                        src_row = src_row * 23 + i_src;
                            valid = valid && (i_src < 23);
                                pf *= sqrt((R) (i_src - 0));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - 1;
                        src_row = src_row * 23 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 1)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0125663706144, 0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - -1;
                        src_row = src_row * 23 + i_src;
                            valid = valid && (i_src < 23);
                                pf *= sqrt((R) (i_src - 0));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(-0.0125663706144, -0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - 1;
                        src_row = src_row * 23 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 2)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - -1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src < 2);
                                pf *= sqrt((R) (i_src - 0));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 23 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 23 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 3)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0125663706144, 0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - -1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src < 2);
                                pf *= sqrt((R) (i_src - 0));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 23 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(-0.0125663706144, -0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 23 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
            s_psik[1-idx][nrow] = H_psi_k_sum;
            psi_out_v_r += H_psi_k_sum.real();
            psi_out_v_i += H_psi_k_sum.imag();

            // swap psi_k and psi_k_next
            idx = 1 - idx;
        }

        psi_out0(t, 0) = psi_out_v_r;
        psi_out0(t, 1) = psi_out_v_i;
    }
}

__global__
void ovlps_grad_kernel0(R *ctrls, R *states, R *ovlp_r, R *ovlp_i, R *d_ovlps_r, R *d_ovlps_i)
{
    const unsigned int ct = 0;
    const unsigned int t = blockIdx.x;
    const unsigned int nstate = blockIdx.y;
    const unsigned int d_nc = blockIdx.z;
    const unsigned int nrow = threadIdx.x;

    int s = ct ? -1 : 1;

    int nrow_cur = nrow;
    const int i_dst0 = nrow_cur % 23;
    nrow_cur /= 23;
    const int i_dst1 = nrow_cur % 2;
    nrow_cur /= 2;

    C psi_out_v = C::complex(psi_out0(t, 0), psi_out0(t, 1));
    C d_psi_out_v = C::complex(0, 0);
        __shared__ C s_psik[2][46];
        __shared__ C s_d_psik[2][46];
        __syncthreads();
        int idx = 0;
        s_psik[0][nrow] = psi_out_v;
        s_d_psik[0][nrow] = d_psi_out_v;

    for (int k = 1; k <= TAYLOR_ORDER; k++) {
        // psi_k -> (pf*H)psi_k
        // psi_out -> psi_out + (pf*H)psi_k
        __syncthreads();

        R cpf, pf;
        C ppf, t_ppf, H_psi_k_sum, d_H_psi_k_sum;
        int i_src, src_row, valid;
        H_psi_k_sum = 0;
        d_H_psi_k_sum = 0;
                cpf = 1.0;
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(0.0, -1.73415914478e-05);
                        ppf *= i_dst0 * i_dst0;
                        ppf *= i_dst1 ;
                    t_ppf += ppf;
                    ppf = C::complex(0.0, -0.774088429845);
                        ppf *= i_dst1 ;
                    t_ppf += ppf;
                    ppf = C::complex(0.0, 0.774088429845);
                        ppf *= i_dst1 * i_dst1;
                    t_ppf += ppf;
                    ppf = C::complex(0.0, 0.0126591104295);
                        ppf *= i_dst0 ;
                        ppf *= i_dst1 ;
                    t_ppf += ppf;
                    ppf = C::complex(0.0, -8.16814089933e-06);
                        ppf *= i_dst0 ;
                    t_ppf += ppf;
                    ppf = C::complex(0.0, 8.16814089933e-06);
                        ppf *= i_dst0 * i_dst0;
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - 0;
                    src_row = src_row * 2 + i_src;
                    i_src = i_dst0 - 0;
                    src_row = src_row * 23 + i_src;
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == -1)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                cpf = ctrls[idxctrls(t, 0)];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(0.0, -0.0125663706144);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - 0;
                    src_row = src_row * 2 + i_src;
                    i_src = i_dst0 - -1;
                    src_row = src_row * 23 + i_src;
                        valid = valid && (i_src < 23);
                            pf *= sqrt((R) (i_src - 0));
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 0)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(0.0, -0.0125663706144);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - 0;
                    src_row = src_row * 2 + i_src;
                    i_src = i_dst0 - 1;
                    src_row = src_row * 23 + i_src;
                        valid = valid && (i_src >= 0);
                            pf *= sqrt((R) (i_src + 1));
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 0)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                cpf = ctrls[idxctrls(t, 1)];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(0.0125663706144, 0.0);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - 0;
                    src_row = src_row * 2 + i_src;
                    i_src = i_dst0 - -1;
                    src_row = src_row * 23 + i_src;
                        valid = valid && (i_src < 23);
                            pf *= sqrt((R) (i_src - 0));
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 1)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(-0.0125663706144, -0.0);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - 0;
                    src_row = src_row * 2 + i_src;
                    i_src = i_dst0 - 1;
                    src_row = src_row * 23 + i_src;
                        valid = valid && (i_src >= 0);
                            pf *= sqrt((R) (i_src + 1));
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 1)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                cpf = ctrls[idxctrls(t, 2)];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(0.0, -0.0125663706144);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - -1;
                    src_row = src_row * 2 + i_src;
                        valid = valid && (i_src < 2);
                            pf *= sqrt((R) (i_src - 0));
                    i_src = i_dst0 - 0;
                    src_row = src_row * 23 + i_src;
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 2)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(0.0, -0.0125663706144);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - 1;
                    src_row = src_row * 2 + i_src;
                        valid = valid && (i_src >= 0);
                            pf *= sqrt((R) (i_src + 1));
                    i_src = i_dst0 - 0;
                    src_row = src_row * 23 + i_src;
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 2)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                cpf = ctrls[idxctrls(t, 3)];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(0.0125663706144, 0.0);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - -1;
                    src_row = src_row * 2 + i_src;
                        valid = valid && (i_src < 2);
                            pf *= sqrt((R) (i_src - 0));
                    i_src = i_dst0 - 0;
                    src_row = src_row * 23 + i_src;
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 3)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(-0.0125663706144, -0.0);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - 1;
                    src_row = src_row * 2 + i_src;
                        valid = valid && (i_src >= 0);
                            pf *= sqrt((R) (i_src + 1));
                    i_src = i_dst0 - 0;
                    src_row = src_row * 23 + i_src;
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 3)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];

        s_psik[1-idx][nrow] = H_psi_k_sum;
        s_d_psik[1-idx][nrow] = d_H_psi_k_sum;

        // swap psi_k and psi_k_next
        idx = 1 - idx;

        psi_out_v += H_psi_k_sum;
        d_psi_out_v += d_H_psi_k_sum;
    }

    // reuse shared memory for calculation of overlaps
    // - for conjugate
    s_psik[idx][nrow] = C::complex(psi_out_ct0(t+1, 0), -psi_out_ct0(t+1, 1)) * psi_out_v;
    s_d_psik[idx][nrow] = C::complex(psi_out_ct0(t+1, 0), -psi_out_ct0(t+1, 1)) * d_psi_out_v;
    __syncthreads();
        if (nrow < 23) {
            s_psik[idx][nrow] += s_psik[idx][nrow + 23];
            s_d_psik[idx][nrow] += s_d_psik[idx][nrow + 23];
        }
        __syncthreads();
        if (nrow < 11) {
            s_psik[idx][nrow] += s_psik[idx][nrow + 12];
            s_d_psik[idx][nrow] += s_d_psik[idx][nrow + 12];
        }
        __syncthreads();
        if (nrow < 6) {
            s_psik[idx][nrow] += s_psik[idx][nrow + 6];
            s_d_psik[idx][nrow] += s_d_psik[idx][nrow + 6];
        }
        __syncthreads();
        if (nrow < 3) {
            s_psik[idx][nrow] += s_psik[idx][nrow + 3];
            s_d_psik[idx][nrow] += s_d_psik[idx][nrow + 3];
        }
        __syncthreads();
        if (nrow < 1) {
            s_psik[idx][nrow] += s_psik[idx][nrow + 2];
            s_d_psik[idx][nrow] += s_d_psik[idx][nrow + 2];
        }
        __syncthreads();
        if (nrow < 1) {
            s_psik[idx][nrow] += s_psik[idx][nrow + 1];
            s_d_psik[idx][nrow] += s_d_psik[idx][nrow + 1];
        }
        __syncthreads();

    if (nrow == 0) {
        if ((t == 0) && (d_nc == 0)) {
            atomicAddD(ovlp_r, s_psik[idx][0].real());
            atomicAddD(ovlp_i, s_psik[idx][0].imag());
        }
        atomicAddD(d_ovlps_r + d_nc + NCTRLS*t, s_d_psik[idx][0].real());
        atomicAddD(d_ovlps_i + d_nc + NCTRLS*t, s_d_psik[idx][0].imag());
    }
}


R *states_d1;
R *ovlp_r_d1;
R *ovlp_i_d1;
R *d_ovlps_r_d1;
R *d_ovlps_i_d1;

__global__
void prop_state_kernel1_noct(R *ctrls, R *states)
{
    const unsigned int ct = 0;
    const unsigned int nstate = blockIdx.y;
    const unsigned int nrow = threadIdx.x;
    __shared__ C s_psik[2][48];
    __syncthreads();
    const short s = ct ? -1 : 1;

    int nrow_cur = nrow;
    const int i_dst0 = nrow_cur % 24;
    nrow_cur /= 24;
    const int i_dst1 = nrow_cur % 2;
    nrow_cur /= 2;

    /* R psi_out_v = psi_out(0, ri); */
    R psi_out_v_r = psi_out1(0, 0);
    R psi_out_v_i = psi_out1(0, 1);
    for (int t = 1; t <= PLEN; t++) {
        int ctrl_t = ct ? (PLEN - t) : (t-1);
        int idx = 0;
        s_psik[0][nrow] = C::complex(psi_out_v_r, psi_out_v_i);

        for (int k = 1; k <= TAYLOR_ORDER; k++) {
            // psi_k -> (pf*H)psi_k
            // psi_out -> psi_out + (pf*H)psi_k
            __syncthreads();

            // TODO: Parallelize this
            R cpf, pf;
            C ppf, t_ppf, H_psi_k_sum;
            int i_src, src_row, valid;
            H_psi_k_sum = 0;
                    cpf = 1.0 / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -1.73415914478e-05);
                            ppf *= i_dst0 * i_dst0;
                            ppf *= i_dst1 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, -0.774088429845);
                            ppf *= i_dst1 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, 0.774088429845);
                            ppf *= i_dst1 * i_dst1;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, 0.0126591104295);
                            ppf *= i_dst0 ;
                            ppf *= i_dst1 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, -8.16814089933e-06);
                            ppf *= i_dst0 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, 8.16814089933e-06);
                            ppf *= i_dst0 * i_dst0;
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - 0;
                        src_row = src_row * 24 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 0)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - -1;
                        src_row = src_row * 24 + i_src;
                            valid = valid && (i_src < 24);
                                pf *= sqrt((R) (i_src - 0));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - 1;
                        src_row = src_row * 24 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 1)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0125663706144, 0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - -1;
                        src_row = src_row * 24 + i_src;
                            valid = valid && (i_src < 24);
                                pf *= sqrt((R) (i_src - 0));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(-0.0125663706144, -0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - 1;
                        src_row = src_row * 24 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 2)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - -1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src < 2);
                                pf *= sqrt((R) (i_src - 0));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 24 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 24 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 3)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0125663706144, 0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - -1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src < 2);
                                pf *= sqrt((R) (i_src - 0));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 24 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(-0.0125663706144, -0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 24 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
            s_psik[1-idx][nrow] = H_psi_k_sum;
            psi_out_v_r += H_psi_k_sum.real();
            psi_out_v_i += H_psi_k_sum.imag();

            // swap psi_k and psi_k_next
            idx = 1 - idx;
        }

        psi_out1(t, 0) = psi_out_v_r;
        psi_out1(t, 1) = psi_out_v_i;
    }
}
__global__
void prop_state_kernel1_withct(R *ctrls, R *states)
{
    const unsigned int ct = 1;
    const unsigned int nstate = blockIdx.y;
    const unsigned int nrow = threadIdx.x;
    __shared__ C s_psik[2][48];
    __syncthreads();
    const short s = ct ? -1 : 1;

    int nrow_cur = nrow;
    const int i_dst0 = nrow_cur % 24;
    nrow_cur /= 24;
    const int i_dst1 = nrow_cur % 2;
    nrow_cur /= 2;

    /* R psi_out_v = psi_out(0, ri); */
    R psi_out_v_r = psi_out1(0, 0);
    R psi_out_v_i = psi_out1(0, 1);
    for (int t = 1; t <= PLEN; t++) {
        int ctrl_t = ct ? (PLEN - t) : (t-1);
        int idx = 0;
        s_psik[0][nrow] = C::complex(psi_out_v_r, psi_out_v_i);

        for (int k = 1; k <= TAYLOR_ORDER; k++) {
            // psi_k -> (pf*H)psi_k
            // psi_out -> psi_out + (pf*H)psi_k
            __syncthreads();

            // TODO: Parallelize this
            R cpf, pf;
            C ppf, t_ppf, H_psi_k_sum;
            int i_src, src_row, valid;
            H_psi_k_sum = 0;
                    cpf = 1.0 / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -1.73415914478e-05);
                            ppf *= i_dst0 * i_dst0;
                            ppf *= i_dst1 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, -0.774088429845);
                            ppf *= i_dst1 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, 0.774088429845);
                            ppf *= i_dst1 * i_dst1;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, 0.0126591104295);
                            ppf *= i_dst0 ;
                            ppf *= i_dst1 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, -8.16814089933e-06);
                            ppf *= i_dst0 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, 8.16814089933e-06);
                            ppf *= i_dst0 * i_dst0;
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - 0;
                        src_row = src_row * 24 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 0)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - -1;
                        src_row = src_row * 24 + i_src;
                            valid = valid && (i_src < 24);
                                pf *= sqrt((R) (i_src - 0));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - 1;
                        src_row = src_row * 24 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 1)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0125663706144, 0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - -1;
                        src_row = src_row * 24 + i_src;
                            valid = valid && (i_src < 24);
                                pf *= sqrt((R) (i_src - 0));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(-0.0125663706144, -0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - 1;
                        src_row = src_row * 24 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 2)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - -1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src < 2);
                                pf *= sqrt((R) (i_src - 0));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 24 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 24 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 3)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0125663706144, 0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - -1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src < 2);
                                pf *= sqrt((R) (i_src - 0));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 24 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(-0.0125663706144, -0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 24 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
            s_psik[1-idx][nrow] = H_psi_k_sum;
            psi_out_v_r += H_psi_k_sum.real();
            psi_out_v_i += H_psi_k_sum.imag();

            // swap psi_k and psi_k_next
            idx = 1 - idx;
        }

        psi_out1(t, 0) = psi_out_v_r;
        psi_out1(t, 1) = psi_out_v_i;
    }
}

__global__
void ovlps_grad_kernel1(R *ctrls, R *states, R *ovlp_r, R *ovlp_i, R *d_ovlps_r, R *d_ovlps_i)
{
    const unsigned int ct = 0;
    const unsigned int t = blockIdx.x;
    const unsigned int nstate = blockIdx.y;
    const unsigned int d_nc = blockIdx.z;
    const unsigned int nrow = threadIdx.x;

    int s = ct ? -1 : 1;

    int nrow_cur = nrow;
    const int i_dst0 = nrow_cur % 24;
    nrow_cur /= 24;
    const int i_dst1 = nrow_cur % 2;
    nrow_cur /= 2;

    C psi_out_v = C::complex(psi_out1(t, 0), psi_out1(t, 1));
    C d_psi_out_v = C::complex(0, 0);
        __shared__ C s_psik[2][48];
        __shared__ C s_d_psik[2][48];
        __syncthreads();
        int idx = 0;
        s_psik[0][nrow] = psi_out_v;
        s_d_psik[0][nrow] = d_psi_out_v;

    for (int k = 1; k <= TAYLOR_ORDER; k++) {
        // psi_k -> (pf*H)psi_k
        // psi_out -> psi_out + (pf*H)psi_k
        __syncthreads();

        R cpf, pf;
        C ppf, t_ppf, H_psi_k_sum, d_H_psi_k_sum;
        int i_src, src_row, valid;
        H_psi_k_sum = 0;
        d_H_psi_k_sum = 0;
                cpf = 1.0;
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(0.0, -1.73415914478e-05);
                        ppf *= i_dst0 * i_dst0;
                        ppf *= i_dst1 ;
                    t_ppf += ppf;
                    ppf = C::complex(0.0, -0.774088429845);
                        ppf *= i_dst1 ;
                    t_ppf += ppf;
                    ppf = C::complex(0.0, 0.774088429845);
                        ppf *= i_dst1 * i_dst1;
                    t_ppf += ppf;
                    ppf = C::complex(0.0, 0.0126591104295);
                        ppf *= i_dst0 ;
                        ppf *= i_dst1 ;
                    t_ppf += ppf;
                    ppf = C::complex(0.0, -8.16814089933e-06);
                        ppf *= i_dst0 ;
                    t_ppf += ppf;
                    ppf = C::complex(0.0, 8.16814089933e-06);
                        ppf *= i_dst0 * i_dst0;
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - 0;
                    src_row = src_row * 2 + i_src;
                    i_src = i_dst0 - 0;
                    src_row = src_row * 24 + i_src;
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == -1)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                cpf = ctrls[idxctrls(t, 0)];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(0.0, -0.0125663706144);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - 0;
                    src_row = src_row * 2 + i_src;
                    i_src = i_dst0 - -1;
                    src_row = src_row * 24 + i_src;
                        valid = valid && (i_src < 24);
                            pf *= sqrt((R) (i_src - 0));
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 0)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(0.0, -0.0125663706144);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - 0;
                    src_row = src_row * 2 + i_src;
                    i_src = i_dst0 - 1;
                    src_row = src_row * 24 + i_src;
                        valid = valid && (i_src >= 0);
                            pf *= sqrt((R) (i_src + 1));
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 0)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                cpf = ctrls[idxctrls(t, 1)];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(0.0125663706144, 0.0);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - 0;
                    src_row = src_row * 2 + i_src;
                    i_src = i_dst0 - -1;
                    src_row = src_row * 24 + i_src;
                        valid = valid && (i_src < 24);
                            pf *= sqrt((R) (i_src - 0));
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 1)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(-0.0125663706144, -0.0);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - 0;
                    src_row = src_row * 2 + i_src;
                    i_src = i_dst0 - 1;
                    src_row = src_row * 24 + i_src;
                        valid = valid && (i_src >= 0);
                            pf *= sqrt((R) (i_src + 1));
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 1)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                cpf = ctrls[idxctrls(t, 2)];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(0.0, -0.0125663706144);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - -1;
                    src_row = src_row * 2 + i_src;
                        valid = valid && (i_src < 2);
                            pf *= sqrt((R) (i_src - 0));
                    i_src = i_dst0 - 0;
                    src_row = src_row * 24 + i_src;
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 2)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(0.0, -0.0125663706144);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - 1;
                    src_row = src_row * 2 + i_src;
                        valid = valid && (i_src >= 0);
                            pf *= sqrt((R) (i_src + 1));
                    i_src = i_dst0 - 0;
                    src_row = src_row * 24 + i_src;
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 2)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                cpf = ctrls[idxctrls(t, 3)];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(0.0125663706144, 0.0);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - -1;
                    src_row = src_row * 2 + i_src;
                        valid = valid && (i_src < 2);
                            pf *= sqrt((R) (i_src - 0));
                    i_src = i_dst0 - 0;
                    src_row = src_row * 24 + i_src;
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 3)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(-0.0125663706144, -0.0);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - 1;
                    src_row = src_row * 2 + i_src;
                        valid = valid && (i_src >= 0);
                            pf *= sqrt((R) (i_src + 1));
                    i_src = i_dst0 - 0;
                    src_row = src_row * 24 + i_src;
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 3)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];

        s_psik[1-idx][nrow] = H_psi_k_sum;
        s_d_psik[1-idx][nrow] = d_H_psi_k_sum;

        // swap psi_k and psi_k_next
        idx = 1 - idx;

        psi_out_v += H_psi_k_sum;
        d_psi_out_v += d_H_psi_k_sum;
    }

    // reuse shared memory for calculation of overlaps
    // - for conjugate
    s_psik[idx][nrow] = C::complex(psi_out_ct1(t+1, 0), -psi_out_ct1(t+1, 1)) * psi_out_v;
    s_d_psik[idx][nrow] = C::complex(psi_out_ct1(t+1, 0), -psi_out_ct1(t+1, 1)) * d_psi_out_v;
    __syncthreads();
        if (nrow < 24) {
            s_psik[idx][nrow] += s_psik[idx][nrow + 24];
            s_d_psik[idx][nrow] += s_d_psik[idx][nrow + 24];
        }
        __syncthreads();
        if (nrow < 12) {
            s_psik[idx][nrow] += s_psik[idx][nrow + 12];
            s_d_psik[idx][nrow] += s_d_psik[idx][nrow + 12];
        }
        __syncthreads();
        if (nrow < 6) {
            s_psik[idx][nrow] += s_psik[idx][nrow + 6];
            s_d_psik[idx][nrow] += s_d_psik[idx][nrow + 6];
        }
        __syncthreads();
        if (nrow < 3) {
            s_psik[idx][nrow] += s_psik[idx][nrow + 3];
            s_d_psik[idx][nrow] += s_d_psik[idx][nrow + 3];
        }
        __syncthreads();
        if (nrow < 1) {
            s_psik[idx][nrow] += s_psik[idx][nrow + 2];
            s_d_psik[idx][nrow] += s_d_psik[idx][nrow + 2];
        }
        __syncthreads();
        if (nrow < 1) {
            s_psik[idx][nrow] += s_psik[idx][nrow + 1];
            s_d_psik[idx][nrow] += s_d_psik[idx][nrow + 1];
        }
        __syncthreads();

    if (nrow == 0) {
        if ((t == 0) && (d_nc == 0)) {
            atomicAddD(ovlp_r, s_psik[idx][0].real());
            atomicAddD(ovlp_i, s_psik[idx][0].imag());
        }
        atomicAddD(d_ovlps_r + d_nc + NCTRLS*t, s_d_psik[idx][0].real());
        atomicAddD(d_ovlps_i + d_nc + NCTRLS*t, s_d_psik[idx][0].imag());
    }
}


R *states_d2;
R *ovlp_r_d2;
R *ovlp_i_d2;
R *d_ovlps_r_d2;
R *d_ovlps_i_d2;

__global__
void prop_state_kernel2_noct(R *ctrls, R *states)
{
    const unsigned int ct = 0;
    const unsigned int nstate = blockIdx.y;
    const unsigned int nrow = threadIdx.x;
    __shared__ C s_psik[2][50];
    __syncthreads();
    const short s = ct ? -1 : 1;

    int nrow_cur = nrow;
    const int i_dst0 = nrow_cur % 25;
    nrow_cur /= 25;
    const int i_dst1 = nrow_cur % 2;
    nrow_cur /= 2;

    /* R psi_out_v = psi_out(0, ri); */
    R psi_out_v_r = psi_out2(0, 0);
    R psi_out_v_i = psi_out2(0, 1);
    for (int t = 1; t <= PLEN; t++) {
        int ctrl_t = ct ? (PLEN - t) : (t-1);
        int idx = 0;
        s_psik[0][nrow] = C::complex(psi_out_v_r, psi_out_v_i);

        for (int k = 1; k <= TAYLOR_ORDER; k++) {
            // psi_k -> (pf*H)psi_k
            // psi_out -> psi_out + (pf*H)psi_k
            __syncthreads();

            // TODO: Parallelize this
            R cpf, pf;
            C ppf, t_ppf, H_psi_k_sum;
            int i_src, src_row, valid;
            H_psi_k_sum = 0;
                    cpf = 1.0 / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -1.73415914478e-05);
                            ppf *= i_dst0 * i_dst0;
                            ppf *= i_dst1 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, -0.774088429845);
                            ppf *= i_dst1 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, 0.774088429845);
                            ppf *= i_dst1 * i_dst1;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, 0.0126591104295);
                            ppf *= i_dst0 ;
                            ppf *= i_dst1 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, -8.16814089933e-06);
                            ppf *= i_dst0 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, 8.16814089933e-06);
                            ppf *= i_dst0 * i_dst0;
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - 0;
                        src_row = src_row * 25 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 0)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - -1;
                        src_row = src_row * 25 + i_src;
                            valid = valid && (i_src < 25);
                                pf *= sqrt((R) (i_src - 0));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - 1;
                        src_row = src_row * 25 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 1)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0125663706144, 0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - -1;
                        src_row = src_row * 25 + i_src;
                            valid = valid && (i_src < 25);
                                pf *= sqrt((R) (i_src - 0));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(-0.0125663706144, -0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - 1;
                        src_row = src_row * 25 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 2)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - -1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src < 2);
                                pf *= sqrt((R) (i_src - 0));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 25 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 25 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 3)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0125663706144, 0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - -1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src < 2);
                                pf *= sqrt((R) (i_src - 0));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 25 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(-0.0125663706144, -0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 25 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
            s_psik[1-idx][nrow] = H_psi_k_sum;
            psi_out_v_r += H_psi_k_sum.real();
            psi_out_v_i += H_psi_k_sum.imag();

            // swap psi_k and psi_k_next
            idx = 1 - idx;
        }

        psi_out2(t, 0) = psi_out_v_r;
        psi_out2(t, 1) = psi_out_v_i;
    }
}
__global__
void prop_state_kernel2_withct(R *ctrls, R *states)
{
    const unsigned int ct = 1;
    const unsigned int nstate = blockIdx.y;
    const unsigned int nrow = threadIdx.x;
    __shared__ C s_psik[2][50];
    __syncthreads();
    const short s = ct ? -1 : 1;

    int nrow_cur = nrow;
    const int i_dst0 = nrow_cur % 25;
    nrow_cur /= 25;
    const int i_dst1 = nrow_cur % 2;
    nrow_cur /= 2;

    /* R psi_out_v = psi_out(0, ri); */
    R psi_out_v_r = psi_out2(0, 0);
    R psi_out_v_i = psi_out2(0, 1);
    for (int t = 1; t <= PLEN; t++) {
        int ctrl_t = ct ? (PLEN - t) : (t-1);
        int idx = 0;
        s_psik[0][nrow] = C::complex(psi_out_v_r, psi_out_v_i);

        for (int k = 1; k <= TAYLOR_ORDER; k++) {
            // psi_k -> (pf*H)psi_k
            // psi_out -> psi_out + (pf*H)psi_k
            __syncthreads();

            // TODO: Parallelize this
            R cpf, pf;
            C ppf, t_ppf, H_psi_k_sum;
            int i_src, src_row, valid;
            H_psi_k_sum = 0;
                    cpf = 1.0 / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -1.73415914478e-05);
                            ppf *= i_dst0 * i_dst0;
                            ppf *= i_dst1 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, -0.774088429845);
                            ppf *= i_dst1 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, 0.774088429845);
                            ppf *= i_dst1 * i_dst1;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, 0.0126591104295);
                            ppf *= i_dst0 ;
                            ppf *= i_dst1 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, -8.16814089933e-06);
                            ppf *= i_dst0 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, 8.16814089933e-06);
                            ppf *= i_dst0 * i_dst0;
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - 0;
                        src_row = src_row * 25 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 0)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - -1;
                        src_row = src_row * 25 + i_src;
                            valid = valid && (i_src < 25);
                                pf *= sqrt((R) (i_src - 0));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - 1;
                        src_row = src_row * 25 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 1)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0125663706144, 0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - -1;
                        src_row = src_row * 25 + i_src;
                            valid = valid && (i_src < 25);
                                pf *= sqrt((R) (i_src - 0));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(-0.0125663706144, -0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - 1;
                        src_row = src_row * 25 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 2)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - -1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src < 2);
                                pf *= sqrt((R) (i_src - 0));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 25 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 25 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 3)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0125663706144, 0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - -1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src < 2);
                                pf *= sqrt((R) (i_src - 0));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 25 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(-0.0125663706144, -0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 25 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
            s_psik[1-idx][nrow] = H_psi_k_sum;
            psi_out_v_r += H_psi_k_sum.real();
            psi_out_v_i += H_psi_k_sum.imag();

            // swap psi_k and psi_k_next
            idx = 1 - idx;
        }

        psi_out2(t, 0) = psi_out_v_r;
        psi_out2(t, 1) = psi_out_v_i;
    }
}

__global__
void ovlps_grad_kernel2(R *ctrls, R *states, R *ovlp_r, R *ovlp_i, R *d_ovlps_r, R *d_ovlps_i)
{
    const unsigned int ct = 0;
    const unsigned int t = blockIdx.x;
    const unsigned int nstate = blockIdx.y;
    const unsigned int d_nc = blockIdx.z;
    const unsigned int nrow = threadIdx.x;

    int s = ct ? -1 : 1;

    int nrow_cur = nrow;
    const int i_dst0 = nrow_cur % 25;
    nrow_cur /= 25;
    const int i_dst1 = nrow_cur % 2;
    nrow_cur /= 2;

    C psi_out_v = C::complex(psi_out2(t, 0), psi_out2(t, 1));
    C d_psi_out_v = C::complex(0, 0);
        __shared__ C s_psik[2][50];
        __shared__ C s_d_psik[2][50];
        __syncthreads();
        int idx = 0;
        s_psik[0][nrow] = psi_out_v;
        s_d_psik[0][nrow] = d_psi_out_v;

    for (int k = 1; k <= TAYLOR_ORDER; k++) {
        // psi_k -> (pf*H)psi_k
        // psi_out -> psi_out + (pf*H)psi_k
        __syncthreads();

        R cpf, pf;
        C ppf, t_ppf, H_psi_k_sum, d_H_psi_k_sum;
        int i_src, src_row, valid;
        H_psi_k_sum = 0;
        d_H_psi_k_sum = 0;
                cpf = 1.0;
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(0.0, -1.73415914478e-05);
                        ppf *= i_dst0 * i_dst0;
                        ppf *= i_dst1 ;
                    t_ppf += ppf;
                    ppf = C::complex(0.0, -0.774088429845);
                        ppf *= i_dst1 ;
                    t_ppf += ppf;
                    ppf = C::complex(0.0, 0.774088429845);
                        ppf *= i_dst1 * i_dst1;
                    t_ppf += ppf;
                    ppf = C::complex(0.0, 0.0126591104295);
                        ppf *= i_dst0 ;
                        ppf *= i_dst1 ;
                    t_ppf += ppf;
                    ppf = C::complex(0.0, -8.16814089933e-06);
                        ppf *= i_dst0 ;
                    t_ppf += ppf;
                    ppf = C::complex(0.0, 8.16814089933e-06);
                        ppf *= i_dst0 * i_dst0;
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - 0;
                    src_row = src_row * 2 + i_src;
                    i_src = i_dst0 - 0;
                    src_row = src_row * 25 + i_src;
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == -1)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                cpf = ctrls[idxctrls(t, 0)];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(0.0, -0.0125663706144);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - 0;
                    src_row = src_row * 2 + i_src;
                    i_src = i_dst0 - -1;
                    src_row = src_row * 25 + i_src;
                        valid = valid && (i_src < 25);
                            pf *= sqrt((R) (i_src - 0));
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 0)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(0.0, -0.0125663706144);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - 0;
                    src_row = src_row * 2 + i_src;
                    i_src = i_dst0 - 1;
                    src_row = src_row * 25 + i_src;
                        valid = valid && (i_src >= 0);
                            pf *= sqrt((R) (i_src + 1));
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 0)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                cpf = ctrls[idxctrls(t, 1)];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(0.0125663706144, 0.0);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - 0;
                    src_row = src_row * 2 + i_src;
                    i_src = i_dst0 - -1;
                    src_row = src_row * 25 + i_src;
                        valid = valid && (i_src < 25);
                            pf *= sqrt((R) (i_src - 0));
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 1)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(-0.0125663706144, -0.0);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - 0;
                    src_row = src_row * 2 + i_src;
                    i_src = i_dst0 - 1;
                    src_row = src_row * 25 + i_src;
                        valid = valid && (i_src >= 0);
                            pf *= sqrt((R) (i_src + 1));
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 1)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                cpf = ctrls[idxctrls(t, 2)];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(0.0, -0.0125663706144);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - -1;
                    src_row = src_row * 2 + i_src;
                        valid = valid && (i_src < 2);
                            pf *= sqrt((R) (i_src - 0));
                    i_src = i_dst0 - 0;
                    src_row = src_row * 25 + i_src;
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 2)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(0.0, -0.0125663706144);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - 1;
                    src_row = src_row * 2 + i_src;
                        valid = valid && (i_src >= 0);
                            pf *= sqrt((R) (i_src + 1));
                    i_src = i_dst0 - 0;
                    src_row = src_row * 25 + i_src;
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 2)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                cpf = ctrls[idxctrls(t, 3)];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(0.0125663706144, 0.0);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - -1;
                    src_row = src_row * 2 + i_src;
                        valid = valid && (i_src < 2);
                            pf *= sqrt((R) (i_src - 0));
                    i_src = i_dst0 - 0;
                    src_row = src_row * 25 + i_src;
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 3)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(-0.0125663706144, -0.0);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - 1;
                    src_row = src_row * 2 + i_src;
                        valid = valid && (i_src >= 0);
                            pf *= sqrt((R) (i_src + 1));
                    i_src = i_dst0 - 0;
                    src_row = src_row * 25 + i_src;
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 3)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];

        s_psik[1-idx][nrow] = H_psi_k_sum;
        s_d_psik[1-idx][nrow] = d_H_psi_k_sum;

        // swap psi_k and psi_k_next
        idx = 1 - idx;

        psi_out_v += H_psi_k_sum;
        d_psi_out_v += d_H_psi_k_sum;
    }

    // reuse shared memory for calculation of overlaps
    // - for conjugate
    s_psik[idx][nrow] = C::complex(psi_out_ct2(t+1, 0), -psi_out_ct2(t+1, 1)) * psi_out_v;
    s_d_psik[idx][nrow] = C::complex(psi_out_ct2(t+1, 0), -psi_out_ct2(t+1, 1)) * d_psi_out_v;
    __syncthreads();
        if (nrow < 25) {
            s_psik[idx][nrow] += s_psik[idx][nrow + 25];
            s_d_psik[idx][nrow] += s_d_psik[idx][nrow + 25];
        }
        __syncthreads();
        if (nrow < 12) {
            s_psik[idx][nrow] += s_psik[idx][nrow + 13];
            s_d_psik[idx][nrow] += s_d_psik[idx][nrow + 13];
        }
        __syncthreads();
        if (nrow < 6) {
            s_psik[idx][nrow] += s_psik[idx][nrow + 7];
            s_d_psik[idx][nrow] += s_d_psik[idx][nrow + 7];
        }
        __syncthreads();
        if (nrow < 3) {
            s_psik[idx][nrow] += s_psik[idx][nrow + 4];
            s_d_psik[idx][nrow] += s_d_psik[idx][nrow + 4];
        }
        __syncthreads();
        if (nrow < 2) {
            s_psik[idx][nrow] += s_psik[idx][nrow + 2];
            s_d_psik[idx][nrow] += s_d_psik[idx][nrow + 2];
        }
        __syncthreads();
        if (nrow < 1) {
            s_psik[idx][nrow] += s_psik[idx][nrow + 1];
            s_d_psik[idx][nrow] += s_d_psik[idx][nrow + 1];
        }
        __syncthreads();

    if (nrow == 0) {
        if ((t == 0) && (d_nc == 0)) {
            atomicAddD(ovlp_r, s_psik[idx][0].real());
            atomicAddD(ovlp_i, s_psik[idx][0].imag());
        }
        atomicAddD(d_ovlps_r + d_nc + NCTRLS*t, s_d_psik[idx][0].real());
        atomicAddD(d_ovlps_i + d_nc + NCTRLS*t, s_d_psik[idx][0].imag());
    }
}


R *states_d3;
R *ovlp_r_d3;
R *ovlp_i_d3;
R *d_ovlps_r_d3;
R *d_ovlps_i_d3;

__global__
void prop_state_kernel3_noct(R *ctrls, R *states)
{
    const unsigned int ct = 0;
    const unsigned int nstate = blockIdx.y;
    const unsigned int nrow = threadIdx.x;
    __shared__ C s_psik[2][52];
    __syncthreads();
    const short s = ct ? -1 : 1;

    int nrow_cur = nrow;
    const int i_dst0 = nrow_cur % 26;
    nrow_cur /= 26;
    const int i_dst1 = nrow_cur % 2;
    nrow_cur /= 2;

    /* R psi_out_v = psi_out(0, ri); */
    R psi_out_v_r = psi_out3(0, 0);
    R psi_out_v_i = psi_out3(0, 1);
    for (int t = 1; t <= PLEN; t++) {
        int ctrl_t = ct ? (PLEN - t) : (t-1);
        int idx = 0;
        s_psik[0][nrow] = C::complex(psi_out_v_r, psi_out_v_i);

        for (int k = 1; k <= TAYLOR_ORDER; k++) {
            // psi_k -> (pf*H)psi_k
            // psi_out -> psi_out + (pf*H)psi_k
            __syncthreads();

            // TODO: Parallelize this
            R cpf, pf;
            C ppf, t_ppf, H_psi_k_sum;
            int i_src, src_row, valid;
            H_psi_k_sum = 0;
                    cpf = 1.0 / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -1.73415914478e-05);
                            ppf *= i_dst0 * i_dst0;
                            ppf *= i_dst1 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, -0.774088429845);
                            ppf *= i_dst1 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, 0.774088429845);
                            ppf *= i_dst1 * i_dst1;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, 0.0126591104295);
                            ppf *= i_dst0 ;
                            ppf *= i_dst1 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, -8.16814089933e-06);
                            ppf *= i_dst0 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, 8.16814089933e-06);
                            ppf *= i_dst0 * i_dst0;
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - 0;
                        src_row = src_row * 26 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 0)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - -1;
                        src_row = src_row * 26 + i_src;
                            valid = valid && (i_src < 26);
                                pf *= sqrt((R) (i_src - 0));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - 1;
                        src_row = src_row * 26 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 1)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0125663706144, 0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - -1;
                        src_row = src_row * 26 + i_src;
                            valid = valid && (i_src < 26);
                                pf *= sqrt((R) (i_src - 0));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(-0.0125663706144, -0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - 1;
                        src_row = src_row * 26 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 2)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - -1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src < 2);
                                pf *= sqrt((R) (i_src - 0));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 26 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 26 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 3)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0125663706144, 0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - -1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src < 2);
                                pf *= sqrt((R) (i_src - 0));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 26 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(-0.0125663706144, -0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 26 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
            s_psik[1-idx][nrow] = H_psi_k_sum;
            psi_out_v_r += H_psi_k_sum.real();
            psi_out_v_i += H_psi_k_sum.imag();

            // swap psi_k and psi_k_next
            idx = 1 - idx;
        }

        psi_out3(t, 0) = psi_out_v_r;
        psi_out3(t, 1) = psi_out_v_i;
    }
}
__global__
void prop_state_kernel3_withct(R *ctrls, R *states)
{
    const unsigned int ct = 1;
    const unsigned int nstate = blockIdx.y;
    const unsigned int nrow = threadIdx.x;
    __shared__ C s_psik[2][52];
    __syncthreads();
    const short s = ct ? -1 : 1;

    int nrow_cur = nrow;
    const int i_dst0 = nrow_cur % 26;
    nrow_cur /= 26;
    const int i_dst1 = nrow_cur % 2;
    nrow_cur /= 2;

    /* R psi_out_v = psi_out(0, ri); */
    R psi_out_v_r = psi_out3(0, 0);
    R psi_out_v_i = psi_out3(0, 1);
    for (int t = 1; t <= PLEN; t++) {
        int ctrl_t = ct ? (PLEN - t) : (t-1);
        int idx = 0;
        s_psik[0][nrow] = C::complex(psi_out_v_r, psi_out_v_i);

        for (int k = 1; k <= TAYLOR_ORDER; k++) {
            // psi_k -> (pf*H)psi_k
            // psi_out -> psi_out + (pf*H)psi_k
            __syncthreads();

            // TODO: Parallelize this
            R cpf, pf;
            C ppf, t_ppf, H_psi_k_sum;
            int i_src, src_row, valid;
            H_psi_k_sum = 0;
                    cpf = 1.0 / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -1.73415914478e-05);
                            ppf *= i_dst0 * i_dst0;
                            ppf *= i_dst1 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, -0.774088429845);
                            ppf *= i_dst1 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, 0.774088429845);
                            ppf *= i_dst1 * i_dst1;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, 0.0126591104295);
                            ppf *= i_dst0 ;
                            ppf *= i_dst1 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, -8.16814089933e-06);
                            ppf *= i_dst0 ;
                        t_ppf += ppf;
                        ppf = C::complex(0.0, 8.16814089933e-06);
                            ppf *= i_dst0 * i_dst0;
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - 0;
                        src_row = src_row * 26 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 0)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - -1;
                        src_row = src_row * 26 + i_src;
                            valid = valid && (i_src < 26);
                                pf *= sqrt((R) (i_src - 0));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - 1;
                        src_row = src_row * 26 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 1)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0125663706144, 0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - -1;
                        src_row = src_row * 26 + i_src;
                            valid = valid && (i_src < 26);
                                pf *= sqrt((R) (i_src - 0));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(-0.0125663706144, -0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 0;
                        src_row = src_row * 2 + i_src;
                        i_src = i_dst0 - 1;
                        src_row = src_row * 26 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 2)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - -1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src < 2);
                                pf *= sqrt((R) (i_src - 0));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 26 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0, -0.0125663706144);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 26 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    cpf = ctrls[idxctrls(ctrl_t, 3)] / k;
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(0.0125663706144, 0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - -1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src < 2);
                                pf *= sqrt((R) (i_src - 0));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 26 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                    valid = 1;
                    src_row = 0;
                    t_ppf = 0;
                        ppf = C::complex(-0.0125663706144, -0.0);
                        t_ppf += ppf;
                    pf = 1;
                        i_src = i_dst1 - 1;
                        src_row = src_row * 2 + i_src;
                            valid = valid && (i_src >= 0);
                                pf *= sqrt((R) (i_src + 1));
                        i_src = i_dst0 - 0;
                        src_row = src_row * 26 + i_src;
                    src_row *= valid;
                    pf = valid ? pf : 0.0;
                    H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
            s_psik[1-idx][nrow] = H_psi_k_sum;
            psi_out_v_r += H_psi_k_sum.real();
            psi_out_v_i += H_psi_k_sum.imag();

            // swap psi_k and psi_k_next
            idx = 1 - idx;
        }

        psi_out3(t, 0) = psi_out_v_r;
        psi_out3(t, 1) = psi_out_v_i;
    }
}

__global__
void ovlps_grad_kernel3(R *ctrls, R *states, R *ovlp_r, R *ovlp_i, R *d_ovlps_r, R *d_ovlps_i)
{
    const unsigned int ct = 0;
    const unsigned int t = blockIdx.x;
    const unsigned int nstate = blockIdx.y;
    const unsigned int d_nc = blockIdx.z;
    const unsigned int nrow = threadIdx.x;

    int s = ct ? -1 : 1;

    int nrow_cur = nrow;
    const int i_dst0 = nrow_cur % 26;
    nrow_cur /= 26;
    const int i_dst1 = nrow_cur % 2;
    nrow_cur /= 2;

    C psi_out_v = C::complex(psi_out3(t, 0), psi_out3(t, 1));
    C d_psi_out_v = C::complex(0, 0);
        __shared__ C s_psik[2][52];
        __shared__ C s_d_psik[2][52];
        __syncthreads();
        int idx = 0;
        s_psik[0][nrow] = psi_out_v;
        s_d_psik[0][nrow] = d_psi_out_v;

    for (int k = 1; k <= TAYLOR_ORDER; k++) {
        // psi_k -> (pf*H)psi_k
        // psi_out -> psi_out + (pf*H)psi_k
        __syncthreads();

        R cpf, pf;
        C ppf, t_ppf, H_psi_k_sum, d_H_psi_k_sum;
        int i_src, src_row, valid;
        H_psi_k_sum = 0;
        d_H_psi_k_sum = 0;
                cpf = 1.0;
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(0.0, -1.73415914478e-05);
                        ppf *= i_dst0 * i_dst0;
                        ppf *= i_dst1 ;
                    t_ppf += ppf;
                    ppf = C::complex(0.0, -0.774088429845);
                        ppf *= i_dst1 ;
                    t_ppf += ppf;
                    ppf = C::complex(0.0, 0.774088429845);
                        ppf *= i_dst1 * i_dst1;
                    t_ppf += ppf;
                    ppf = C::complex(0.0, 0.0126591104295);
                        ppf *= i_dst0 ;
                        ppf *= i_dst1 ;
                    t_ppf += ppf;
                    ppf = C::complex(0.0, -8.16814089933e-06);
                        ppf *= i_dst0 ;
                    t_ppf += ppf;
                    ppf = C::complex(0.0, 8.16814089933e-06);
                        ppf *= i_dst0 * i_dst0;
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - 0;
                    src_row = src_row * 2 + i_src;
                    i_src = i_dst0 - 0;
                    src_row = src_row * 26 + i_src;
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == -1)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                cpf = ctrls[idxctrls(t, 0)];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(0.0, -0.0125663706144);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - 0;
                    src_row = src_row * 2 + i_src;
                    i_src = i_dst0 - -1;
                    src_row = src_row * 26 + i_src;
                        valid = valid && (i_src < 26);
                            pf *= sqrt((R) (i_src - 0));
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 0)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(0.0, -0.0125663706144);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - 0;
                    src_row = src_row * 2 + i_src;
                    i_src = i_dst0 - 1;
                    src_row = src_row * 26 + i_src;
                        valid = valid && (i_src >= 0);
                            pf *= sqrt((R) (i_src + 1));
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 0)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                cpf = ctrls[idxctrls(t, 1)];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(0.0125663706144, 0.0);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - 0;
                    src_row = src_row * 2 + i_src;
                    i_src = i_dst0 - -1;
                    src_row = src_row * 26 + i_src;
                        valid = valid && (i_src < 26);
                            pf *= sqrt((R) (i_src - 0));
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 1)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(-0.0125663706144, -0.0);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - 0;
                    src_row = src_row * 2 + i_src;
                    i_src = i_dst0 - 1;
                    src_row = src_row * 26 + i_src;
                        valid = valid && (i_src >= 0);
                            pf *= sqrt((R) (i_src + 1));
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 1)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                cpf = ctrls[idxctrls(t, 2)];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(0.0, -0.0125663706144);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - -1;
                    src_row = src_row * 2 + i_src;
                        valid = valid && (i_src < 2);
                            pf *= sqrt((R) (i_src - 0));
                    i_src = i_dst0 - 0;
                    src_row = src_row * 26 + i_src;
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 2)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(0.0, -0.0125663706144);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - 1;
                    src_row = src_row * 2 + i_src;
                        valid = valid && (i_src >= 0);
                            pf *= sqrt((R) (i_src + 1));
                    i_src = i_dst0 - 0;
                    src_row = src_row * 26 + i_src;
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 2)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                cpf = ctrls[idxctrls(t, 3)];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(0.0125663706144, 0.0);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - -1;
                    src_row = src_row * 2 + i_src;
                        valid = valid && (i_src < 2);
                            pf *= sqrt((R) (i_src - 0));
                    i_src = i_dst0 - 0;
                    src_row = src_row * 26 + i_src;
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 3)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];
                valid = 1;
                src_row = 0;
                t_ppf = 0;
                    ppf = C::complex(-0.0125663706144, -0.0);
                    t_ppf += ppf;
                pf = 1.0 / k;
                    i_src = i_dst1 - 1;
                    src_row = src_row * 2 + i_src;
                        valid = valid && (i_src >= 0);
                            pf *= sqrt((R) (i_src + 1));
                    i_src = i_dst0 - 0;
                    src_row = src_row * 26 + i_src;
                src_row *= valid;
                pf = valid ? pf : 0.0;
                H_psi_k_sum += (s * cpf * pf) * t_ppf * s_psik[idx][src_row];
                d_H_psi_k_sum += (s * cpf * pf) * t_ppf * s_d_psik[idx][src_row];
                if (d_nc == 3)
                    d_H_psi_k_sum += (s * pf) * t_ppf * s_psik[idx][src_row];

        s_psik[1-idx][nrow] = H_psi_k_sum;
        s_d_psik[1-idx][nrow] = d_H_psi_k_sum;

        // swap psi_k and psi_k_next
        idx = 1 - idx;

        psi_out_v += H_psi_k_sum;
        d_psi_out_v += d_H_psi_k_sum;
    }

    // reuse shared memory for calculation of overlaps
    // - for conjugate
    s_psik[idx][nrow] = C::complex(psi_out_ct3(t+1, 0), -psi_out_ct3(t+1, 1)) * psi_out_v;
    s_d_psik[idx][nrow] = C::complex(psi_out_ct3(t+1, 0), -psi_out_ct3(t+1, 1)) * d_psi_out_v;
    __syncthreads();
        if (nrow < 26) {
            s_psik[idx][nrow] += s_psik[idx][nrow + 26];
            s_d_psik[idx][nrow] += s_d_psik[idx][nrow + 26];
        }
        __syncthreads();
        if (nrow < 13) {
            s_psik[idx][nrow] += s_psik[idx][nrow + 13];
            s_d_psik[idx][nrow] += s_d_psik[idx][nrow + 13];
        }
        __syncthreads();
        if (nrow < 6) {
            s_psik[idx][nrow] += s_psik[idx][nrow + 7];
            s_d_psik[idx][nrow] += s_d_psik[idx][nrow + 7];
        }
        __syncthreads();
        if (nrow < 3) {
            s_psik[idx][nrow] += s_psik[idx][nrow + 4];
            s_d_psik[idx][nrow] += s_d_psik[idx][nrow + 4];
        }
        __syncthreads();
        if (nrow < 2) {
            s_psik[idx][nrow] += s_psik[idx][nrow + 2];
            s_d_psik[idx][nrow] += s_d_psik[idx][nrow + 2];
        }
        __syncthreads();
        if (nrow < 1) {
            s_psik[idx][nrow] += s_psik[idx][nrow + 1];
            s_d_psik[idx][nrow] += s_d_psik[idx][nrow + 1];
        }
        __syncthreads();

    if (nrow == 0) {
        if ((t == 0) && (d_nc == 0)) {
            atomicAddD(ovlp_r, s_psik[idx][0].real());
            atomicAddD(ovlp_i, s_psik[idx][0].imag());
        }
        atomicAddD(d_ovlps_r + d_nc + NCTRLS*t, s_d_psik[idx][0].real());
        atomicAddD(d_ovlps_i + d_nc + NCTRLS*t, s_d_psik[idx][0].imag());
    }
}



void grape_step(R *ctrls, R *ovlp_r, R *ovlp_i, R *d_ovlp_r, R *d_ovlp_i)
{
    gpuErrchk(hipMemcpy(ctrls_d, ctrls, NCTRLS*PLEN*sizeof(R), hipMemcpyHostToDevice));
    hipStream_t stream0;
    hipStreamCreate(&stream0);
    hipStream_t stream1;
    hipStreamCreate(&stream1);
    hipStream_t stream2;
    hipStreamCreate(&stream2);
    hipStream_t stream3;
    hipStreamCreate(&stream3);
    dim3 blocks(1, NSTATE, 1);
    dim3 blocks_ovlp(PLEN, NSTATE, NCTRLS);
    hipMemsetAsync(ovlp_r_d0, 0, sizeof(R), stream0);
    hipMemsetAsync(ovlp_i_d0, 0, sizeof(R), stream0);
    hipMemsetAsync(d_ovlps_r_d0, 0, ctrls_size*sizeof(R), stream0);
    hipMemsetAsync(d_ovlps_i_d0, 0, ctrls_size*sizeof(R), stream0);
    hipMemsetAsync(ovlp_r_d1, 0, sizeof(R), stream1);
    hipMemsetAsync(ovlp_i_d1, 0, sizeof(R), stream1);
    hipMemsetAsync(d_ovlps_r_d1, 0, ctrls_size*sizeof(R), stream1);
    hipMemsetAsync(d_ovlps_i_d1, 0, ctrls_size*sizeof(R), stream1);
    hipMemsetAsync(ovlp_r_d2, 0, sizeof(R), stream2);
    hipMemsetAsync(ovlp_i_d2, 0, sizeof(R), stream2);
    hipMemsetAsync(d_ovlps_r_d2, 0, ctrls_size*sizeof(R), stream2);
    hipMemsetAsync(d_ovlps_i_d2, 0, ctrls_size*sizeof(R), stream2);
    hipMemsetAsync(ovlp_r_d3, 0, sizeof(R), stream3);
    hipMemsetAsync(ovlp_i_d3, 0, sizeof(R), stream3);
    hipMemsetAsync(d_ovlps_r_d3, 0, ctrls_size*sizeof(R), stream3);
    hipMemsetAsync(d_ovlps_i_d3, 0, ctrls_size*sizeof(R), stream3);
    dim3 threads0(46, 1, 1);
    prop_state_kernel0_noct<<<blocks, threads0, 0, stream0>>>(ctrls_d, states_d0);
    prop_state_kernel0_withct<<<blocks, threads0, 0, stream0>>>(ctrls_d, states_d0);
    ovlps_grad_kernel0<<<blocks_ovlp, threads0, 0, stream0>>>(
        ctrls_d, states_d0, ovlp_r_d0, ovlp_i_d0,
        d_ovlps_r_d0, d_ovlps_i_d0
    );
    dim3 threads1(48, 1, 1);
    prop_state_kernel1_noct<<<blocks, threads1, 0, stream1>>>(ctrls_d, states_d1);
    prop_state_kernel1_withct<<<blocks, threads1, 0, stream1>>>(ctrls_d, states_d1);
    ovlps_grad_kernel1<<<blocks_ovlp, threads1, 0, stream1>>>(
        ctrls_d, states_d1, ovlp_r_d1, ovlp_i_d1,
        d_ovlps_r_d1, d_ovlps_i_d1
    );
    dim3 threads2(50, 1, 1);
    prop_state_kernel2_noct<<<blocks, threads2, 0, stream2>>>(ctrls_d, states_d2);
    prop_state_kernel2_withct<<<blocks, threads2, 0, stream2>>>(ctrls_d, states_d2);
    ovlps_grad_kernel2<<<blocks_ovlp, threads2, 0, stream2>>>(
        ctrls_d, states_d2, ovlp_r_d2, ovlp_i_d2,
        d_ovlps_r_d2, d_ovlps_i_d2
    );
    dim3 threads3(52, 1, 1);
    prop_state_kernel3_noct<<<blocks, threads3, 0, stream3>>>(ctrls_d, states_d3);
    prop_state_kernel3_withct<<<blocks, threads3, 0, stream3>>>(ctrls_d, states_d3);
    ovlps_grad_kernel3<<<blocks_ovlp, threads3, 0, stream3>>>(
        ctrls_d, states_d3, ovlp_r_d3, ovlp_i_d3,
        d_ovlps_r_d3, d_ovlps_i_d3
    );
    hipMemcpyAsync(d_ovlp_r + 0*ctrls_size, d_ovlps_r_d0, ctrls_size*sizeof(R),
                    hipMemcpyDeviceToHost, stream0);
    hipMemcpyAsync(d_ovlp_i + 0*ctrls_size, d_ovlps_i_d0, ctrls_size*sizeof(R),
                    hipMemcpyDeviceToHost, stream0);
    hipMemcpyAsync(ovlp_r + 0, ovlp_r_d0, sizeof(R), hipMemcpyDeviceToHost, stream0);
    hipMemcpyAsync(ovlp_i + 0, ovlp_i_d0, sizeof(R), hipMemcpyDeviceToHost, stream0);
    hipMemcpyAsync(d_ovlp_r + 1*ctrls_size, d_ovlps_r_d1, ctrls_size*sizeof(R),
                    hipMemcpyDeviceToHost, stream1);
    hipMemcpyAsync(d_ovlp_i + 1*ctrls_size, d_ovlps_i_d1, ctrls_size*sizeof(R),
                    hipMemcpyDeviceToHost, stream1);
    hipMemcpyAsync(ovlp_r + 1, ovlp_r_d1, sizeof(R), hipMemcpyDeviceToHost, stream1);
    hipMemcpyAsync(ovlp_i + 1, ovlp_i_d1, sizeof(R), hipMemcpyDeviceToHost, stream1);
    hipMemcpyAsync(d_ovlp_r + 2*ctrls_size, d_ovlps_r_d2, ctrls_size*sizeof(R),
                    hipMemcpyDeviceToHost, stream2);
    hipMemcpyAsync(d_ovlp_i + 2*ctrls_size, d_ovlps_i_d2, ctrls_size*sizeof(R),
                    hipMemcpyDeviceToHost, stream2);
    hipMemcpyAsync(ovlp_r + 2, ovlp_r_d2, sizeof(R), hipMemcpyDeviceToHost, stream2);
    hipMemcpyAsync(ovlp_i + 2, ovlp_i_d2, sizeof(R), hipMemcpyDeviceToHost, stream2);
    hipMemcpyAsync(d_ovlp_r + 3*ctrls_size, d_ovlps_r_d3, ctrls_size*sizeof(R),
                    hipMemcpyDeviceToHost, stream3);
    hipMemcpyAsync(d_ovlp_i + 3*ctrls_size, d_ovlps_i_d3, ctrls_size*sizeof(R),
                    hipMemcpyDeviceToHost, stream3);
    hipMemcpyAsync(ovlp_r + 3, ovlp_r_d3, sizeof(R), hipMemcpyDeviceToHost, stream3);
    hipMemcpyAsync(ovlp_i + 3, ovlp_i_d3, sizeof(R), hipMemcpyDeviceToHost, stream3);
    hipDeviceSynchronize(); gpuErrchk(hipGetLastError());
    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);
}

void init_gpu_memory()
{
    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
    gpuErrchk(hipMalloc(&ctrls_d, ctrls_size * sizeof(R)));

    int states_size;
    states_size = 2*NSTATE*(PLEN+1)*46*2;
    gpuErrchk(hipMalloc(&states_d0, states_size * sizeof(R)));
    gpuErrchk(hipMemset(states_d0, 0, states_size*sizeof(R)));

    gpuErrchk(hipMalloc(&ovlp_r_d0, sizeof(R)));
    gpuErrchk(hipMalloc(&ovlp_i_d0, sizeof(R)));

    gpuErrchk(hipMalloc(&d_ovlps_r_d0, ctrls_size * sizeof(R)));
    gpuErrchk(hipMemset(d_ovlps_r_d0, 0, ctrls_size*sizeof(R)));
    gpuErrchk(hipMalloc(&d_ovlps_i_d0, ctrls_size * sizeof(R)));
    gpuErrchk(hipMemset(d_ovlps_i_d0, 0, ctrls_size*sizeof(R)));
    states_size = 2*NSTATE*(PLEN+1)*48*2;
    gpuErrchk(hipMalloc(&states_d1, states_size * sizeof(R)));
    gpuErrchk(hipMemset(states_d1, 0, states_size*sizeof(R)));

    gpuErrchk(hipMalloc(&ovlp_r_d1, sizeof(R)));
    gpuErrchk(hipMalloc(&ovlp_i_d1, sizeof(R)));

    gpuErrchk(hipMalloc(&d_ovlps_r_d1, ctrls_size * sizeof(R)));
    gpuErrchk(hipMemset(d_ovlps_r_d1, 0, ctrls_size*sizeof(R)));
    gpuErrchk(hipMalloc(&d_ovlps_i_d1, ctrls_size * sizeof(R)));
    gpuErrchk(hipMemset(d_ovlps_i_d1, 0, ctrls_size*sizeof(R)));
    states_size = 2*NSTATE*(PLEN+1)*50*2;
    gpuErrchk(hipMalloc(&states_d2, states_size * sizeof(R)));
    gpuErrchk(hipMemset(states_d2, 0, states_size*sizeof(R)));

    gpuErrchk(hipMalloc(&ovlp_r_d2, sizeof(R)));
    gpuErrchk(hipMalloc(&ovlp_i_d2, sizeof(R)));

    gpuErrchk(hipMalloc(&d_ovlps_r_d2, ctrls_size * sizeof(R)));
    gpuErrchk(hipMemset(d_ovlps_r_d2, 0, ctrls_size*sizeof(R)));
    gpuErrchk(hipMalloc(&d_ovlps_i_d2, ctrls_size * sizeof(R)));
    gpuErrchk(hipMemset(d_ovlps_i_d2, 0, ctrls_size*sizeof(R)));
    states_size = 2*NSTATE*(PLEN+1)*52*2;
    gpuErrchk(hipMalloc(&states_d3, states_size * sizeof(R)));
    gpuErrchk(hipMemset(states_d3, 0, states_size*sizeof(R)));

    gpuErrchk(hipMalloc(&ovlp_r_d3, sizeof(R)));
    gpuErrchk(hipMalloc(&ovlp_i_d3, sizeof(R)));

    gpuErrchk(hipMalloc(&d_ovlps_r_d3, ctrls_size * sizeof(R)));
    gpuErrchk(hipMemset(d_ovlps_r_d3, 0, ctrls_size*sizeof(R)));
    gpuErrchk(hipMalloc(&d_ovlps_i_d3, ctrls_size * sizeof(R)));
    gpuErrchk(hipMemset(d_ovlps_i_d3, 0, ctrls_size*sizeof(R)));
}

void load_states(int nvar, R *psi0, R *psif)
{
    int states_size;
    switch (nvar) {
        case 0:
            states_size = 2*NSTATE*(PLEN+1)*46*2;
            gpuErrchk(hipMemset(states_d0, 0, states_size*sizeof(R)));
            for (int i = 0; i < NSTATE; i++) {
                gpuErrchk(hipMemcpy(
                    states_d0 + idxstates0(0, i, 0, 0, 0),
                    psi0 + (2*i*46), 2*46*sizeof(R), hipMemcpyHostToDevice)
                );
                gpuErrchk(hipMemcpy(
                    states_d0 + idxstates0(1, i, 0, 0, 0),
                    psif + (2*i*46), 2*46*sizeof(R), hipMemcpyHostToDevice)
                );
            }
            break;
        case 1:
            states_size = 2*NSTATE*(PLEN+1)*48*2;
            gpuErrchk(hipMemset(states_d1, 0, states_size*sizeof(R)));
            for (int i = 0; i < NSTATE; i++) {
                gpuErrchk(hipMemcpy(
                    states_d1 + idxstates1(0, i, 0, 0, 0),
                    psi0 + (2*i*48), 2*48*sizeof(R), hipMemcpyHostToDevice)
                );
                gpuErrchk(hipMemcpy(
                    states_d1 + idxstates1(1, i, 0, 0, 0),
                    psif + (2*i*48), 2*48*sizeof(R), hipMemcpyHostToDevice)
                );
            }
            break;
        case 2:
            states_size = 2*NSTATE*(PLEN+1)*50*2;
            gpuErrchk(hipMemset(states_d2, 0, states_size*sizeof(R)));
            for (int i = 0; i < NSTATE; i++) {
                gpuErrchk(hipMemcpy(
                    states_d2 + idxstates2(0, i, 0, 0, 0),
                    psi0 + (2*i*50), 2*50*sizeof(R), hipMemcpyHostToDevice)
                );
                gpuErrchk(hipMemcpy(
                    states_d2 + idxstates2(1, i, 0, 0, 0),
                    psif + (2*i*50), 2*50*sizeof(R), hipMemcpyHostToDevice)
                );
            }
            break;
        case 3:
            states_size = 2*NSTATE*(PLEN+1)*52*2;
            gpuErrchk(hipMemset(states_d3, 0, states_size*sizeof(R)));
            for (int i = 0; i < NSTATE; i++) {
                gpuErrchk(hipMemcpy(
                    states_d3 + idxstates3(0, i, 0, 0, 0),
                    psi0 + (2*i*52), 2*52*sizeof(R), hipMemcpyHostToDevice)
                );
                gpuErrchk(hipMemcpy(
                    states_d3 + idxstates3(1, i, 0, 0, 0),
                    psif + (2*i*52), 2*52*sizeof(R), hipMemcpyHostToDevice)
                );
            }
            break;
    }
}

void get_states(int nvar, R *states)
{
    int states_size;
    switch (nvar) {
        case 0:
            states_size = 2*NSTATE*(PLEN+1)*46*2;
            gpuErrchk(hipMemcpy(states, states_d0, states_size*sizeof(R), hipMemcpyDeviceToHost));
            break;
        case 1:
            states_size = 2*NSTATE*(PLEN+1)*48*2;
            gpuErrchk(hipMemcpy(states, states_d1, states_size*sizeof(R), hipMemcpyDeviceToHost));
            break;
        case 2:
            states_size = 2*NSTATE*(PLEN+1)*50*2;
            gpuErrchk(hipMemcpy(states, states_d2, states_size*sizeof(R), hipMemcpyDeviceToHost));
            break;
        case 3:
            states_size = 2*NSTATE*(PLEN+1)*52*2;
            gpuErrchk(hipMemcpy(states, states_d3, states_size*sizeof(R), hipMemcpyDeviceToHost));
            break;
    }
}